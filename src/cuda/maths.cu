#include "hip/hip_runtime.h"
#include "../../include/cudart.h"

__device__ double v_scal(struct v3d a, struct v3d b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ double v_norm(struct v3d a) {
    return sqrt(v_scal(a, a));
}



__device__ struct v3d v_sub(struct v3d a, struct v3d b) {
    return (struct v3d) {
        .x = a.x - b.x,
        .y = a.y - b.y,
        .z = a.z - b.z
    };
}
__device__ struct v3d v_add(struct v3d a, struct v3d b) {
    return (struct v3d) {
        .x = a.x + b.x,
        .y = a.y + b.y,
        .z = a.z + b.z
    };
}
__device__ struct v3d v_mul(struct v3d a, double b) {
    return (struct v3d) {
        .x = a.x * b,
        .y = a.y * b,
        .z = a.z * b
    };
}

__device__ struct v3d v_normalize(struct v3d v) {
    float magnitude = sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
    return (struct v3d) {
        .x = v.x / magnitude,
        .y = v.y / magnitude,
        .z = v.z / magnitude
    };
}

struct v3d v_cross(struct v3d v1,struct v3d v2) {
    struct v3d result;
    result.x = v1.y * v2.z - v1.z * v2.y;
    result.y = v1.z * v2.x - v1.x * v2.z;
    result.z = v1.x * v2.y - v1.y * v2.x;
    return result;
}