#include "hip/hip_runtime.h"
#include "../../include/cuda_trace_functions.hpp"

__device__ struct v3d ray_intersect_sphere(struct ray ray, struct sphere sphere) {

    struct v3d oc = v_sub(ray.origin, sphere.center);
    float a = v_scal(ray.direction, ray.direction);
    float b = 2.0f * v_scal(oc, ray.direction);
    float c = v_scal(oc, oc) - sphere.radius * sphere.radius;
    float discriminant = b * b - 4.0f * a * c;

    struct v3d no_intersection = {INFINITY, INFINITY, INFINITY};

    if (discriminant < 0) {
        return no_intersection;
    } else {
        // Use the smaller positive solution, if available
        float t0 = (-b - sqrt(discriminant)) / (2.0f * a);
        float t1 = (-b + sqrt(discriminant)) / (2.0f * a);
        
        float t;
        if (t0 > 0) {
            t = t0;
        } else if (t1 > 0) {
            t = t1;
        } else {
            return no_intersection;
        }

        struct v3d intersection_point = v_add(ray.origin, v_mul(ray.direction, t));
        return intersection_point;
    }
}

__device__ struct v3d get_closest_intersection(struct ray ray, struct sphere** closest_sphere) {
    if (closest_sphere && *closest_sphere) {
        *closest_sphere = NULL;
    }
    struct v3d closest_intersection = {INFINITY, INFINITY, INFINITY};
    float closest_distance = INFINITY;

    for (int i = 0; i < sphere_count; i++) {
        //dbg(5,"Checking collision with sphere at {%f,%f,%f}",spheres[i].center.x,spheres[i].center.y,spheres[i].center.z);
        struct v3d intersection_point = ray_intersect_sphere((struct ray) ray, spheres[i]);
        
        if (intersection_point.x != INFINITY) {
            //dbg(5,"Collision with sphere %d",i);
            float distance = v_norm(v_sub(intersection_point, ray.origin));
            
            if (distance < closest_distance) {
                if (closest_sphere) *closest_sphere = &spheres[i];
                closest_intersection = intersection_point;
                closest_distance = distance;
            }
        }
    }

    return closest_intersection;
} 