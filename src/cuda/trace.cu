#include "hip/hip_runtime.h"
#include "SDL2/SDL_stdinc.h"
#include "../../include/cudart.h"
#include "../../include/cuda_trace_functions.hpp"



__device__ struct sphere spheres[MAX_SPHERES]; 
__device__ int sphere_count;
__device__ struct camera cam;

__device__ void prandint(int* state) {
    *state = *state * 1103515245 + 12345;
    *state = (*state / 65536) % 32768;

}



__device__ struct v3d get_normal(struct v3d intersection_point, struct sphere* sphere) {
    struct v3d normal = v_sub(intersection_point, sphere->center);
    normal = v_normalize(normal);
    return normal;
}

__device__ Uint32 trace(struct ray ray) {
    
    struct sphere* closest_sphere = NULL;
    struct v3d closest_intersection = get_closest_intersection(ray, &closest_sphere);

    if (closest_sphere == NULL) {
        return 0x000000;
    } else if (closest_sphere->material.light_source) {
        return closest_sphere->material.color;
    }

    float color_intensity = 0.0f;

    struct v3d normal = get_normal(closest_intersection, closest_sphere);

    struct v3d light_position = (struct v3d) {0, 0, 0};
    for (int i = 0; i < sphere_count; i++) {
        if (spheres[i].material.light_source) {
            light_position = spheres[i].center;
        }

        // calculate diffuse lighting
        struct v3d light_direction = v_sub(light_position, closest_intersection);
        light_direction = v_normalize(light_direction);

        float diffuse = v_scal(normal, light_direction);
        if (diffuse > 0) {
            color_intensity += diffuse;
        }

        // calculate specular lighting

        struct v3d reflection = v_sub(v_mul(normal, 2 * v_scal(normal, light_direction)), light_direction);
        reflection = v_normalize(reflection);

        struct v3d view_direction = v_sub(cam.position, closest_intersection);
        view_direction = v_normalize(view_direction);

        float specular = pow(v_scal(reflection, view_direction), closest_sphere->material.gloss);
        if (specular > 0) {
            color_intensity += specular;
        }

        // calculate shadows



    }



    if (color_intensity > 1.0f) {
        color_intensity = 1.0f;
    }

    Uint8 r,g,b;
    get_pixel_rgb(closest_sphere->material.color, &r, &g,&b);

    r = r * color_intensity;
    g = g * color_intensity;
    b = b * color_intensity;

    Uint32 final_color;
    set_pixel_rgb(&final_color, r, g, b);
    return final_color;
}

__global__ void cuda_kernel(Uint32* pixels,int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    float px = (2.0f * ((x + 0.5f) / (float)width) - 1.0f) * tanf(FOV * 0.5f) * ASPECT_RATIO;
    float py = (1.0f - 2.0f * ((y + 0.5f) / (float)height)) * tanf(FOV * 0.5f);

    struct v3d ray_direction = v_normalize(v_add(v_add(v_mul(cam.right, px), v_mul(cam.up, py)), cam.forward));

    // Set the pixel at (x, y) based on the ray tracing result
    pixels[y * width + x] = trace((struct ray) {
        .origin = cam.position,
        .direction = ray_direction
    });
}

void launch_tracer(Uint32 pixels[SCREEN_HEIGHT][SCREEN_WIDTH], struct camera l_cam,struct sphere* l_spheres, int l_sphere_count) {

    // Copy the camera and sphere data to the GPU
    hipMemcpyToSymbol(HIP_SYMBOL(cam), &l_cam, sizeof(struct camera));
    hipMemcpyToSymbol(HIP_SYMBOL(spheres), l_spheres, sizeof(struct sphere) * l_sphere_count);
    hipMemcpyToSymbol(HIP_SYMBOL(sphere_count), &l_sphere_count, sizeof(int));
    
    // Allocate memory on the GPU for the pixel buffer
    Uint32* d_pixels;
    size_t bufferSize = sizeof(Uint32) * SCREEN_HEIGHT * SCREEN_WIDTH;
    hipMalloc(&d_pixels, bufferSize);

    // Define the block and grid dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((SCREEN_WIDTH + blockDim.x - 1) / blockDim.x, (SCREEN_HEIGHT + blockDim.y - 1) / blockDim.y);

    // Launch the CUDA kernel
    cuda_kernel<<<gridDim, blockDim>>>(d_pixels,SCREEN_WIDTH, SCREEN_HEIGHT);

    // Copy the pixel data from the GPU to the host memory
    hipMemcpy(pixels, d_pixels, bufferSize, hipMemcpyDeviceToHost);

    // Free the allocated memory on the GPU
    hipFree(d_pixels);

    // Make sure the kernel execution is complete before returning
    hipDeviceSynchronize();
}
